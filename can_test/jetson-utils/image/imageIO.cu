#include "hip/hip_runtime.h"
#include "imageIO.h"
#include "cudaUtility.h"

template<typename T>
__global__ void gpuDetectionOverlayBox_pat( T* input, T* output, int imgWidth, int imgHeight, T* pattern, int x0, int y0, int boxWidth, int boxHeight, const float pat_alpha ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight )
		return;

	const T px_in = pattern[ box_y * boxWidth + box_x ];
	if( px_in.w !=0 )
	{
		if( pat_alpha == 0)
		{
			const T img_in = input[ y * imgWidth + x ];
			output[y * imgWidth + x] = img_in;
		}
		else if( pat_alpha == 255)
			output[y * imgWidth + x] = px_in;
		else
		{
			const float alpha = pat_alpha / 255.0f;
			const float ialph = 1.0f - alpha;
			const T img_in = input[ y * imgWidth + x ];
			output[y * imgWidth + x] = make_float4( alpha * px_in.x + ialph * img_in.x, 
					alpha * px_in.y + ialph * img_in.y,
					alpha * px_in.z + ialph * img_in.z,
					img_in.w );
		}
	}
}

hipError_t cudaDetectionOverlay_pat( float4* input, float4* output, uint32_t width, uint32_t height, float4* pattern, uint32_t x0, uint32_t y0, uint32_t pat_width, uint32_t pat_height, const float alpha)
{
	if( !input || !output || input != output || width == 0 || height == 0 || !pattern || pat_width == 0 || pat_height == 0 )
		return hipErrorInvalidValue;

	const int boxWidth = (int)pat_width;
	const int boxHeight = (int)pat_height;

			// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));
	gpuDetectionOverlayBox_pat<float4><<<gridDim, blockDim>>>(input, output, width, height, pattern, x0, y0, boxWidth, boxHeight, alpha);
	return hipGetLastError(); 
}

template<typename T>
__global__ void gpuDetectionOverlay_pat_selfalpha( T* input, T* output, int imgWidth, int imgHeight, T* pattern, int x0, int y0, int boxWidth, int boxHeight ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight )
		return;

	const T px_in = pattern[ box_y * boxWidth + box_x ];
	if( px_in.w !=0 )
	{

		if( px_in.w == 255)
			output[y * imgWidth + x] = px_in;
		else
		{
			const float alpha = px_in.w / 255.0f;
			const float ialph = 1.0f - alpha;
			const T img_in = input[ y * imgWidth + x ];
			output[y * imgWidth + x] = make_float4( alpha * px_in.x + ialph * img_in.x, 
					alpha * px_in.y + ialph * img_in.y,
					alpha * px_in.z + ialph * img_in.z,
					img_in.w );
		}
	}
}

hipError_t cudaDetectionOverlay_pat_selfalpha( float4* input, float4* output, uint32_t width, uint32_t height, float4* pattern, uint32_t x0, uint32_t y0, uint32_t pat_width, uint32_t pat_height)
{
	if( !input || !output || input != output || width == 0 || height == 0 || !pattern || pat_width == 0 || pat_height == 0 )
		return hipErrorInvalidValue;

	const int boxWidth = (int)pat_width;
	const int boxHeight = (int)pat_height;

			// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));
	gpuDetectionOverlay_pat_selfalpha<float4><<<gridDim, blockDim>>>(input, output, width, height, pattern, x0, y0, boxWidth, boxHeight);
	return hipGetLastError(); 
}

template<typename T>
__global__ void gpuDetectionOverlay_all( T* input, T* output, int imgWidth, int imgHeight, const float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= imgWidth || y >= imgHeight )
		return;

	const T px_in = input[ y * imgWidth + x ];



	if( color.w == 255)
		output[y * imgWidth + x] = color;
	else
	{
		const float alpha = color.w / 255.0f;
		const float ialph = 1.0f - alpha;
		output[y * imgWidth + x] = make_float4( alpha * color.x + ialph * px_in.x, 
					alpha * color.y + ialph * px_in.y,
					alpha * color.z + ialph * px_in.z,
					px_in.w );
	}
}

hipError_t cudaDetectionOverlay_all( float4* input, float4* output, uint32_t width, uint32_t height, const float4 color)
{
	if( !input || !output || input != output || width == 0 || height == 0 )
		return hipErrorInvalidValue;

			// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));
	gpuDetectionOverlay_all<float4><<<gridDim, blockDim>>>(input, output, width, height, color);
	return hipGetLastError(); 
}
